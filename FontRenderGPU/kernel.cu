#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include <windows.h>
#include <opencv2/core/types_c.h>
using namespace cv;

using namespace cooperative_groups;
using namespace std;

hipError_t sumWithCuda(int* sum, const int* arr, unsigned int size);
hipError_t bitmapMatch(int* total_overlap, const char* patch, const char* bitmap, unsigned int patchWidth, unsigned int patchHeight);
hipError_t multiMapMatch(int* total_overlap, const char* image, const char* bitmaps,
	unsigned int n_bitmaps, unsigned int patchWidth, unsigned int patchHeight, unsigned int patchCountX, unsigned int patchCountY);

__global__ void matchKernel(int* total_overlap, const char* patch, const char* bitmap, unsigned int patchHeight)
{
	int x = threadIdx.x;
	int overlap = 0;

	for (int y = 0; y < patchHeight; y++) {
		char bitmap_val = bitmap[y * blockDim.x + x];
		char patch_val = patch[y * blockDim.x + x];
		overlap += bitmap_val == patch_val;
	}

#define FULL_MASK 0xffffffff
	for (int offset = 16; offset > 0; offset /= 2) {
		overlap += __shfl_down_sync(FULL_MASK, overlap, offset);
	}

	if (x == 0) {
		*total_overlap = overlap;
	}
}

// total_overlap[patch_idx, bitmap]
__global__ void multiMatchKernel(int* total_overlap, const char* image, const char* bitmaps,
	unsigned int n_bitmaps, unsigned int patchWidth, unsigned int patchHeight, unsigned int patchCountX, unsigned int patchCountY)
{
	int patchCount = patchCountX * patchCountY;
	int patchSize = patchWidth * patchHeight;
	int x = threadIdx.x;
	int bitmapi = blockIdx.y;
	//char* color_counts = new char[n_colors];
	//__shared__ char main_color[2];
	extern __shared__ char bitmap[];


	for (int y = 0; y < patchHeight; y++) {
		bitmap[y * patchWidth + x] = bitmaps[bitmapi * patchSize + y * patchWidth + x];
	}

	for (int patchi = blockIdx.x; patchi < patchCount; patchi += gridDim.x) {
		int overlap = 0;
		int patchx = patchi % patchCountX;
		int patchy = patchi / patchCountX;

		for (int y = 0; y < patchHeight; y++) {
			char bitmap_val = bitmap[y * patchWidth + x];
			//char bitmap_val = bitmaps[bitmapi * patchSize + y * patchWidth + x];
			char patch_val = image[(patchy * patchHeight + y) * (patchCountX * patchWidth) + (patchx * patchWidth + x)];
			overlap += bitmap_val == patch_val;
		}

#define FULL_MASK 0xffffffff
		for (int offset = 16; offset > 0; offset /= 2) {
			overlap += __shfl_down_sync(FULL_MASK, overlap, offset);
		}

		if (x == 0) {
			total_overlap[patchi * n_bitmaps + bitmapi] = overlap;
		}
	}
}



__global__ void sumKernel(int* sum, const int* arr)
{
	int i = threadIdx.x;
	int val = arr[i];
#define FULL_MASK 0xffffffff
	for (int offset = 16; offset > 0; offset /= 2) {
		val += __shfl_down_sync(FULL_MASK, val, offset);
	}
	if (i == 0) {
		*sum = val;
	}
}


//void memory_test(vector<int>& arr) {
//    arr.resize(10);
//    arr[7] = 2000;
//
//    char* test_bytes = new char[4]{ 1,2,3,4 };
//
//    //memcpy(&arr[0], test_bytes, 3);
//    copy(test_bytes, test_bytes + 3, arr.begin());
//
//    cout << "test:";
//    for (int x : arr) {
//        cout << x << " ";
//    }
//    cout << "\n";
//}


static void load_font_bitmaps(string font_file, int width, int height, vector<char>& bitmaps, vector<int>& unicode_vals, int& num_chars) {
	string numline;
	string bytesline;
	ifstream myfile(font_file);
	int size = width * height;
	//wcout << wchar_t(33) << "\n";
	//wcout << wchar_t(9608) << "\n";
	//wcout << wchar_t(33) << "\n";

	if (myfile.is_open())
	{
		char* int_bytes = new char[4];

		myfile.read(int_bytes, 4);
		std::memcpy(&num_chars, int_bytes, 4);

		bitmaps.resize(size * num_chars);
		unicode_vals.resize(num_chars);

		cout << num_chars << "\n";

		for (int char_idx = 0; char_idx < num_chars; char_idx++) {
			myfile.read(int_bytes, 4);
			std::memcpy(&unicode_vals[char_idx], int_bytes, 4);

			//cout << unicode_val << "\n";

			myfile.read(&bitmaps[size * char_idx], size);
		}
		myfile.close();
	}

	else cout << "Unable to open file";
}

static void render(vector<char>& bitmaps, int width, int height, int index) {
	int offset = width * height * index;
	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			int val = int(bitmaps[offset + y * width + x]);
			cout << (val ? "0" : ".");
		}
		cout << "\n";
	}
	cout << "\n";
}

void render_mat(Mat mat) {

	for (int y = 0; y < mat.rows; y++) {
		for (int x = 0; x < mat.cols; x++) {
			int val = int(mat.data[y * mat.cols + x]);
			cout << (val ? "0" : ".");
		}
		cout << "\n";
	}
	cout << "\n";
}


int main() {
	//cout<< 

	// --- SET UP OUTPUT STREAMS

	std::locale::global(std::locale(""));

	std::cin.imbue(std::locale());
	std::cout.imbue(std::locale());
	std::cerr.imbue(std::locale());
	std::wcin.imbue(std::locale());
	std::wcout.imbue(std::locale());
	std::wcerr.imbue(std::locale());


	// --- LOAD CHARS

	int patchWidth = 32;
	int patchHeight = 57;
	int patchSize = patchWidth * patchHeight;

	string font_file = "font_32-127_32x57";

	vector<char> bitmaps;
	vector<int> unicode_val;
	int num_chars;
	load_font_bitmaps(font_file, patchWidth, patchHeight, bitmaps, unicode_val, num_chars);
	cout << "Read " << num_chars << " chars\n";

	//render(bitmaps, patchWidth, patchHeight, 0);
	//render(bitmaps, patchWidth, patchHeight, 1);
	//render(bitmaps, patchWidth, patchHeight, 2);
	//render(bitmaps, patchWidth, patchHeight, 8585-32);
	//render(bitmaps, patchWidth, patchHeight, 8585 - 33);
	//render(bitmaps, patchWidth, patchHeight, 8585 - 31);
	//render(bitmaps, patchWidth, patchHeight, 9617);
	//render(bitmaps, patchWidth, patchHeight, 9605);

	//for (int i = 0; i < num_chars; i++) {
	//	cout << i << "\n";
	//	cout << unicode_val[i] << "\n";
	//	wchar_t c = wchar_t(unicode_val[i]);
	//	wcout << c << "\n";
	//	//render(bitmaps, patchWidth, patchHeight, i);
	//}
	// 
	// 
	char* bitmaps_arr = new char[bitmaps.size()];
	std::copy(bitmaps.begin(), bitmaps.end(), bitmaps_arr);


	// --- LOAD IMAGE
	String imageName("C:/Users/aaron/Pictures/textrender/tree.jpg");
	Mat src, scaled_img, final_img;
	src = imread(samples::findFile(imageName), IMREAD_COLOR);

	Mat data;
	src.convertTo(data, CV_32F);
	data = data.reshape(1, data.total());
	int n_colors = 14;

	// do kmeans
	Mat labels, centers;
	kmeans(data, n_colors, labels, TermCriteria(CV_TERMCRIT_ITER, 10, 1.0), 3,
		KMEANS_PP_CENTERS, centers);
	cout << labels.rows << "x" << labels.cols << "\n";
	//cout << centers.rows << " " << centers.cols << "\n";
	centers = centers.reshape(3, centers.rows);
	labels.convertTo(labels, CV_8U);
	centers.convertTo(centers, CV_8U);

	for (int i = 0; i < n_colors; i++) {
		cout << centers.at<Vec3b>(i) << "\n"; //<< centers.at<char>(0, 1) << " " << centers.at<char>(0, 2) << "\n";
	}

	cout << src.rows << "x" << src.cols << "\n";
	labels=labels.reshape(0,src.rows);
	cout << labels.rows << "x" << labels.cols << "\n";

	double upscale = 5;
	resize(labels, scaled_img, Size(src.cols * upscale, src.rows * upscale));

	int imgWidth = scaled_img.cols;
	int imgHeight = scaled_img.rows;

	int patchCountX = imgWidth / patchWidth;
	int patchCountY = imgHeight / patchHeight;
	int patchCount = patchCountX * patchCountY;
	//render_mat(scaled_img);
	Rect cropRegion(0, 0, patchCountX * patchWidth, patchCountY * patchHeight);


	Mat roiImage = Mat(scaled_img, cropRegion);
	roiImage.copyTo(final_img);
	//render_mat(final_image);

	//
	
	// UPDATE COLOR TABLE
	HANDLE  hConsole;
	hConsole = GetStdHandle(STD_OUTPUT_HANDLE);

	CONSOLE_SCREEN_BUFFER_INFOEX info;
	info.cbSize = sizeof(CONSOLE_SCREEN_BUFFER_INFOEX);

	GetConsoleScreenBufferInfoEx(hConsole, &info);

	for (int i = 0; i < n_colors; i++) {
		Vec3b col = centers.at<Vec3b>(i);
		info.ColorTable[i + 2] = RGB(col[2], col[1], col[0]);
	}

	SetConsoleScreenBufferInfoEx(hConsole, &info);

	//centers = centers.reshape(3, centers.rows);
	//labels = labels.reshape(1, src.rows);


	//src_gray = imread(imageName, IMREAD_GRAYSCALE);
	//if (src_gray.empty())
	//{
	//	cout << "Cannot read the image: " << imageName << std::endl;
	//	return -1;
	//}
	//
	////cvtColor(src, src_gray, COLOR_BGR2GRAY); // Convert the image to Gray
	//threshold(src_gray, src_binary, 0, 255, 1 + THRESH_OTSU);
	//cout << src_binary.type() << " " << src_binary.channels() << "\n";
	
	


	char* img_arr = new char[labels.total()];
	memcpy(img_arr, labels.data, labels.total());
	cout << "---\n";
	//cout << final_img.total() << "\n";
	//cout << patchCount * patchSize << "\n";
	cout << patchCountX << "x" << patchCountY << "\n";
	cout << patchWidth * patchCountX << "?=" << labels.cols << "\n";
	cout << patchHeight * patchCountY << "?=" << labels.rows << "\n";
	cout << patchSize * patchCount << "?=" << labels.total() << "\n";
	cout << patchSize * num_chars << "?=" << bitmaps.size() << "\n";
	cout << "---\n";

	//char* img_arr = new char[bitmaps.size()];
	//std::copy(bitmaps.begin(), bitmaps.end(), img_arr);

	//wchar_t c = wchar_t(225);
	//wcout << "---" << endl;
	//wcout << c << endl;
	//wcout << "---" << endl;

	// --- CALCULATE OVERLAPS

	int* totalOverlap = new int[patchCount * num_chars];
	hipError_t cudaStatus = multiMapMatch(totalOverlap, img_arr, bitmaps_arr, num_chars, patchWidth, patchHeight, patchCountX, patchCountY);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// --- DISPLAY RESULT


	wofstream myfile;
	myfile.imbue(std::locale());
	myfile.open("output.txt");
	for (int patchi = 0; patchi < patchCount; patchi++) {
		int bestMapi = 0;
		int bestOverlap = 0;
		for (int mapi = 0; mapi < num_chars; mapi++) {
			int overlap = totalOverlap[patchi * num_chars + mapi];
			if (overlap > bestOverlap) {
				bestMapi = mapi;
				bestOverlap = overlap;
			}
		}
		//cout << patchi << " " << bestMapi << " " << bestOverlap << "\n";
		WORD attr = 0;


		bool r = (rand() % 2);
		bool g = (rand() % 2);
		bool b = (rand() % 2);
		bool fi = (rand() % 2);
		attr += b * FOREGROUND_BLUE;
		attr += r * FOREGROUND_RED;
		attr += g * FOREGROUND_GREEN;
		attr += FOREGROUND_INTENSITY;
		//attr += (1-b) * BACKGROUND_BLUE;
		//attr += (1-r) * BACKGROUND_RED;
		//attr += (1-g) * BACKGROUND_GREEN;
		//SetConsoleTextAttribute(hConsole, attr);

		wchar_t c = wchar_t(unicode_val[bestMapi]);
		myfile << c;
		wcout << c;

		if ((patchi + 1) % patchCountX == 0) {
			myfile << "\n";
			wcout << "\n";
		}
	}
	myfile.close();

	//cout << *totalOverlap << "\n";


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t bitmapMatch(int* total_overlap, const char* patch, const char* bitmap, unsigned int patchWidth, unsigned int patchHeight)
{
	char* dev_patch = 0;
	char* dev_bitmap = 0;
	int* dev_total_overlap = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_patch, patchWidth * patchHeight * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_bitmap, patchWidth * patchHeight * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_total_overlap, 1 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_patch, patch, patchWidth * patchHeight * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 1 failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_bitmap, bitmap, patchWidth * patchHeight * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 2 failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	matchKernel << <1, patchWidth >> > (dev_total_overlap, dev_patch, dev_bitmap, patchHeight);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(total_overlap, dev_total_overlap, 1 * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "return hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_total_overlap);
	hipFree(dev_patch);
	hipFree(dev_bitmap);

	return cudaStatus;
}

hipError_t multiMapMatch(int* total_overlap, const char* image, const char* bitmaps,
	unsigned int n_bitmaps, unsigned int patchWidth, unsigned int patchHeight, unsigned int patchCountX, unsigned int patchCountY)
{
	char* dev_image = 0;
	char* dev_bitmaps = 0;
	int* dev_total_overlap = 0;
	hipError_t cudaStatus;
	int patchSize = patchWidth * patchHeight;
	int patchCount = patchCountX * patchCountY;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_image, patchSize * patchCount * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_bitmaps, patchSize * n_bitmaps * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_total_overlap, n_bitmaps * patchCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_image, image, patchSize * patchCount * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 1 failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_bitmaps, bitmaps, patchSize * n_bitmaps * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 2 failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	multiMatchKernel << <dim3(1000, n_bitmaps), patchWidth, patchSize >> > (dev_total_overlap, dev_image, dev_bitmaps, n_bitmaps, patchWidth, patchHeight, patchCountX, patchCountY);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(total_overlap, dev_total_overlap, n_bitmaps * patchCount * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "return hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_total_overlap);
	hipFree(dev_image);
	hipFree(dev_bitmaps);

	return cudaStatus;
}
